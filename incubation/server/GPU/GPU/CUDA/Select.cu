
#include <hip/hip_runtime.h>
extern "C"

// Es werden mehrer Methoden ben�tigt um alle eingaben auszuwerten

__global__ void Integer (double *predict, double *input, double *output) 
{

	// Das Predikat auswerten, dazu schauen ob es mehr als ein Predikat existtiert 
	
	// Die Inputdaten pr�fen auf Dimension f�r mehr als ein Predikat
	
	
	
	if(input[0]<predict[0]){
	
		output[0]= 1;
	
	}else{
		
		output[0] = -1;
	}
	
	//printf("%d \n %d",input[0],output[0]);
	
	__syncthreads();
	
}

__global__ void Double (double *predict, double *input, double *output) 
{

	// Das Predikat auswerten, dazu schauen ob es mehr als ein Predikat existtiert 
	
	// Die Inputdaten pr�fen auf Dimension f�r mehr als ein Predikat
	
	
	if(input[0]<predict[0]){
	
		output[0]= 1;
	
	}else{
		
		output[0] = -1;
	}
	
	//printf("%d \n %d",input[0],output[0]);
	
	__syncthreads();
	
}
